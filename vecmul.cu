#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


__global__ void vectorMultiply(float *A, float *B, float *C, int numElements) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i < numElements) C[i] = A[i] * B[i];
}

int main(void) {
    // params
    float eps = 0.00001;
    int numElements = 50000;
    size_t size = numElements * sizeof(float);

    printf("[Vector addition of %d elements]\n", numElements);

    // allocate host vectors
    printf("allocate host vectors\n");
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // initialize host vectors
    printf("initializing host vectors\n");
    for (int i=0; i<numElements; i++) {
        h_A[i] = i + 1;
        h_B[i] = 1.0 / (i + 1 + eps);
    }

    // allocate device vectors
    printf("allocate device vectors\n");
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    hipError_t err1 = hipMalloc((void **)&d_A, size);
    printf("allocate device vectors pt 1\n");
    hipError_t err2 = hipMalloc((void **)&d_B, size);
    printf("allocate device vectors pt 2\n");
    hipError_t err3 = hipMalloc((void **)&d_C, size);
    printf("allocate device vectors pt 3\n");
    if ((err1 != hipSuccess) || (err2 != hipSuccess) || (err3 != hipSuccess)) {
        fprintf(stderr, "Failed to allocate one of the device vectors\n");
        exit(EXIT_FAILURE);
    }

    // copy host input vectors A and B
    printf("copying host vectors to device\n");
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // launch the kernel
    printf("launching kernel\n");
    int threadsPerBlock = 256;
    int blocksPerGrid = ceil(numElements / (float)threadsPerBlock);
    vectorMultiply <<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // copy device result back to host
    printf("copying device result back to host\n");
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // verify that result is correct by computing on cpu
    printf("verifying results\n");
    for (int i = 0; i<numElements; i++) {
        if (fabs((h_A[i] * h_B[i]) - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    // free device global memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    // free host memory
    free(h_A); free(h_B); free(h_C);

    return 0;
}